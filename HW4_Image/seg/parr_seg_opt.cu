#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "/home/opc/imaging/parr/HW4_Image/mnist/include/mnist/mnist_reader.hpp"

__global__ void thresholdingKernel(uint8_t *images, int numImages, int imageSize, uint8_t threshold) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numImages * imageSize) {
        images[idx] = (images[idx] > threshold) ? 255 : 0;  // 基于阈值的分割
    }
}

int main() {
    auto dataset = mnist::read_dataset<std::vector, std::vector, uint8_t, uint8_t>("..");

    int imageSize = dataset.training_images[0].size();
    int totalImagesSize = dataset.training_images.size() * imageSize;

    uint8_t *h_all_images = new uint8_t[totalImagesSize];
    for (int i = 0; i < dataset.training_images.size(); i++) {
        memcpy(h_all_images + i * imageSize, dataset.training_images[i].data(), imageSize);
    }

    uint8_t *d_images;
    hipMalloc(&d_images, totalImagesSize * sizeof(uint8_t));
    hipMemcpy(d_images, h_all_images, totalImagesSize * sizeof(uint8_t), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (totalImagesSize + threadsPerBlock - 1) / threadsPerBlock;
    uint8_t threshold = 128;

    // Define CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, NULL);

    thresholdingKernel<<<blocks, threadsPerBlock>>>(d_images, dataset.training_images.size(), imageSize, threshold);

    // Record the stop event
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    hipMemcpy(h_all_images, d_images, totalImagesSize * sizeof(uint8_t), hipMemcpyDeviceToHost);

    for (int i = 0; i < dataset.training_images.size(); i++) {
        memcpy(dataset.training_images[i].data(), h_all_images + i * imageSize, imageSize);
    }

    hipFree(d_images);
    delete[] h_all_images;

    // Clean up the events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // ... 你可以保存处理后的图像或进行其他操作 ...

    return 0;
}
