#include "hip/hip_runtime.h"

#include <iostream>
#include "elas.h"
#include "elas_gpu.h"
#include "image.h"

using namespace std;


// Global functions
void process (const char* file_1,const char* file_2);

// Enable profiling
#define PROFILE

int main(int argc, char** argv) {

  // Startup the GPU device
  // https://devtalk.nvidia.com/default/topic/895513/cuda-programming-and-performance/cudamalloc-slow/post/4724457/#4724457
  hipFree(0);

  // Process example frames
  process("../input/cones_left.pgm",   "../input/cones_right.pgm");
  process("../input/aloe_left.pgm",    "../input/aloe_right.pgm");
  process("../input/raindeer_left.pgm","../input/raindeer_right.pgm");
  process("../input/urban1_left.pgm",  "../input/urban1_right.pgm");
  process("../input/urban2_left.pgm",  "../input/urban2_right.pgm");
  process("../input/urban3_left.pgm",  "../input/urban3_right.pgm");
  process("../input/urban4_left.pgm",  "../input/urban4_right.pgm");
  cout << "... done!" << endl;
  
  // Done!
  return EXIT_SUCCESS;
}



/**
 * Compute disparities of pgm image input pair file_1, file_2
 */
void process (const char* file_1,const char* file_2) {

  cout << "Processing: " << file_1 << ", " << file_2 << endl;

  // load images
  image<uchar> *I1,*I2;
  I1 = loadPGM(file_1);
  I2 = loadPGM(file_2);

  // check for correct size
  if (I1->width()<=0 || I1->height() <=0 || I2->width()<=0 || I2->height() <=0 ||
      I1->width()!=I2->width() || I1->height()!=I2->height()) {
    cout << "ERROR: Images must be of same size, but" << endl;
    cout << "       I1: " << I1->width() <<  " x " << I1->height() << 
                 ", I2: " << I2->width() <<  " x " << I2->height() << endl;
    delete I1;
    delete I2;
    return;    
  }

  // get image width and height
  int32_t width  = I1->width();
  int32_t height = I1->height();

  // allocate memory for disparity images
  const int32_t dims[3] = {width,height,width}; // bytes per line = width
  float* D1_data = (float*)malloc(width*height*sizeof(float));
  float* D2_data = (float*)malloc(width*height*sizeof(float));

  // process
  Elas::parameters param;
  param.postprocess_only_left = false;
  //param.subsampling = true;
  ElasGPU elas(param);
  elas.process(I1->data,I2->data,D1_data,D2_data,dims);

  // find maximum disparity for scaling output disparity images to [0..255]
  float disp_max = 0;
  for (int32_t i=0; i<width*height; i++) {
    if (D1_data[i]>disp_max) disp_max = D1_data[i];
    if (D2_data[i]>disp_max) disp_max = D2_data[i];
  }

  // copy float to uchar
  image<uchar> *D1 = new image<uchar>(width,height);
  image<uchar> *D2 = new image<uchar>(width,height);
  for (int32_t i=0; i<width*height; i++) {
    D1->data[i] = (uint8_t)max(255.0*D1_data[i]/disp_max,0.0);
    D2->data[i] = (uint8_t)max(255.0*D2_data[i]/disp_max,0.0);
  }

  // save disparity images
  char output_1[1024];
  char output_2[1024];
  strncpy(output_1,file_1,strlen(file_1)-4);
  strncpy(output_2,file_2,strlen(file_2)-4);
  output_1[strlen(file_1)-4] = '\0';
  output_2[strlen(file_2)-4] = '\0';
  strcat(output_1,"_disp.pgm");
  strcat(output_2,"_disp.pgm");
  savePGM(D1,output_1);
  savePGM(D2,output_2);

  // free memory
  delete I1;
  delete I2;
  delete D1;
  delete D2;
  free(D1_data);
  free(D2_data);
}