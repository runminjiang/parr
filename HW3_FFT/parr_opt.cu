#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define N 512
#define THREADS_PER_BLOCK 256

__global__ void butterflyStepKernelOptimized2(hipfftComplex *a, int n, bool invert) {
    __shared__ hipfftComplex shared_data[THREADS_PER_BLOCK];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= n/2) return;

    int opposite = tid + n/2;
    
    float ang = 2 * PI * tid / n * (invert ? -1 : 1);
    hipfftComplex w = make_hipComplex(__cosf(ang), __sinf(ang)); // 使用快速数学函数

    shared_data[threadIdx.x] = a[tid];
    __syncthreads();

    // 展开蝶形操作的循环
    hipfftComplex a0 = shared_data[threadIdx.x];
    hipfftComplex a1 = a[opposite];
    
    a[tid] = hipCaddf(a0, hipCmulf(w, a1));
    a[opposite] = hipCsubf(a0, hipCmulf(w, a1));
}

void parallelFFTOptimized2(hipfftComplex *data, int n, bool invert) {
    hipSetDevice(0);
    
    hipfftComplex *d_data;
    hipMalloc((void**)&d_data, sizeof(hipfftComplex) * n);
    hipMemcpyAsync(d_data, data, sizeof(hipfftComplex) * n, hipMemcpyHostToDevice); // 使用异步内存传输

    int numBlocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    for(int len = 2; len <= n; len <<= 1) {
        butterflyStepKernelOptimized2<<<numBlocks, THREADS_PER_BLOCK>>>(d_data, len, invert);
    }

    hipMemcpyAsync(data, d_data, sizeof(hipfftComplex) * n, hipMemcpyDeviceToHost); // 使用异步内存传输
    hipFree(d_data);
}
