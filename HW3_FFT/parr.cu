#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define N 512 // 定义适当的大小

__global__ void butterflyStepKernel(hipfftComplex *a, int n, bool invert) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= n/2) return;
    
    int opposite = tid + n/2;
    float ang = 2 * PI * tid / n * (invert ? -1 : 1);
    hipfftComplex w = make_hipComplex(cos(ang), sin(ang));
    
    hipfftComplex a0 = a[tid];
    hipfftComplex a1 = a[opposite];
    
    a[tid] = hipCaddf(a0, hipCmulf(w, a1));
    a[opposite] = hipCsubf(a0, hipCmulf(w, a1));
}

void parallelFFT(hipfftComplex *data, int n, bool invert) {
    // 指定使用第一块GPU
    hipSetDevice(0);
    
    hipfftComplex *d_data;
    hipMalloc((void**)&d_data, sizeof(hipfftComplex) * n);
    hipMemcpy(d_data, data, sizeof(hipfftComplex) * n, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    for(int len = 2; len <= n; len <<= 1) {
        butterflyStepKernel<<<numBlocks, blockSize>>>(d_data, len, invert);
    }

    hipMemcpy(data, d_data, sizeof(hipfftComplex) * n, hipMemcpyDeviceToHost);
    hipFree(d_data);
}
