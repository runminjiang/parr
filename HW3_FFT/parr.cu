#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <iostream>

const float PI = 3.14159265358979323846f;

#define N 512

__global__ void butterflyStepKernel(hipfftComplex *a, int n, bool invert) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= n/2) return;
    
    int opposite = tid + n/2;
    float ang = 2 * PI * tid / n * (invert ? -1 : 1);
    hipfftComplex w = make_hipComplex(cos(ang), sin(ang));
    
    hipfftComplex a0 = a[tid];
    hipfftComplex a1 = a[opposite];
    
    a[tid] = hipCaddf(a0, hipCmulf(w, a1));
    a[opposite] = hipCsubf(a0, hipCmulf(w, a1));
}

void parallelFFT(hipfftComplex *data, int n, bool invert) {
    hipSetDevice(0);
    
    hipfftComplex *d_data;
    hipMalloc((void**)&d_data, sizeof(hipfftComplex) * n);
    hipMemcpy(d_data, data, sizeof(hipfftComplex) * n, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    for(int len = 2; len <= n; len <<= 1) {
        butterflyStepKernel<<<numBlocks, blockSize>>>(d_data, len, invert);
    }

    hipMemcpy(data, d_data, sizeof(hipfftComplex) * n, hipMemcpyDeviceToHost);
    hipFree(d_data);
}

int main() {
    int n = N; // Number of points
    hipfftComplex *data = new hipfftComplex[n];

    // Initialize the data here if needed

    parallelFFT(data, n, false);

    // Display results or further processing

    delete[] data;
    return 0;
}
