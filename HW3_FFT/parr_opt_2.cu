#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

#define N 512
#define THREADS_PER_BLOCK 256
const float PI = 3.14159265358979323846f;

__global__ void butterflyStepKernelOptimized3(hipfftComplex *a, int n, bool invert) {
    __shared__ hipfftComplex shared_data[THREADS_PER_BLOCK];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= n/2) return;

    int opposite = tid + n/2;
    
    float ang = 2 * PI * tid / n * (invert ? -1 : 1);
    hipfftComplex w = make_hipComplex(__cosf(ang), __sinf(ang));

    shared_data[threadIdx.x] = a[tid];
    __syncthreads();

    hipfftComplex a0 = shared_data[threadIdx.x];
    hipfftComplex a1 = a[opposite];
    
    a[tid] = hipCaddf(a0, hipCmulf(w, a1));
    a[opposite] = hipCsubf(a0, hipCmulf(w, a1));
}

void parallelFFTOptimized3(hipfftComplex *data, int n, bool invert) {
    hipSetDevice(0);
    
    hipfftComplex *d_data;
    hipMalloc((void**)&d_data, sizeof(hipfftComplex) * n);
    hipMemcpyAsync(d_data, data, sizeof(hipfftComplex) * n, hipMemcpyHostToDevice);

    int numBlocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    for(int len = 2; len <= n; len <<= 1) {
        butterflyStepKernelOptimized3<<<numBlocks, THREADS_PER_BLOCK>>>(d_data, len, invert);
    }

    hipMemcpyAsync(data, d_data, sizeof(hipfftComplex) * n, hipMemcpyDeviceToHost);
    hipFree(d_data);
}

int main() {
    int n = N;
    hipfftComplex *data = new hipfftComplex[n];

    // Initialize the data (e.g. with random values or some test pattern)
    for(int i = 0; i < n; i++) {
        data[i].x = (float)i;
        data[i].y = 0.0f;
    }

    parallelFFTOptimized3(data, n, false);

    // Optional: Display results or further processing
    for(int i = 0; i < n; i++) {
        std::cout << "(" << data[i].x << ", " << data[i].y << ")" << std::endl;
    }

    delete[] data;
    return 0;
}
